#include <chrono>
#include <iostream>
#include <tuple>

#include "benchmark/benchmark.h"
#include "include/gpu_gemm.cuh"
#include "include/gpu_gemm_entry.cuh"
#include "include/internal/utils.cuh"

static void BM_GpuGemmNaive(benchmark::State &state) {
  const auto N = state.range(0);

  // init input matrices for this benchmark size N
  auto A = fwgpu::Matrix<float>(N, N, 1.5f);
  auto B = fwgpu::Matrix<float>(N, N, 1.5f);

  // allocate device buffers
  auto dptrs = fwgpu::internal::alloc_gemm_mats_on_gpu<float>(A, B);
  float *d_A = std::get<0>(dptrs);
  float *d_B = std::get<1>(dptrs);
  float *d_C = std::get<2>(dptrs);

  float milliseconds = 0.0;
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // loop over benchmark for this size
  dim3 threads(16, 16);
  dim3 blocks((N - 1) / 16 + 1, (N - 1) / 16 + 1);
  for (auto _ : state) {
    hipEventRecord(start);
    fwgpu::gpu_gemm_naive<float><<<blocks, threads>>>(N, N, N, d_A, d_B, d_C);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);
    state.SetIterationTime(milliseconds / 1000);
  }

  double flops_per_itr = 2 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  // free device buffers
  fwgpu::internal::dealloc_gemm_mats_on_gpu<float>(dptrs);
}
BENCHMARK(BM_GpuGemmNaive)->RangeMultiplier(2)->Range(64, 4096)->UseManualTime();

static void BM_CublasSgemm(benchmark::State &state) {
  const auto N = state.range(0);

  // init input matrices for this benchmark size N
  auto A = fwgpu::Matrix<float>(N, N, 1.5f);
  auto B = fwgpu::Matrix<float>(N, N, 1.5f);

  // allocate device buffers
  auto dptrs = fwgpu::internal::alloc_gemm_mats_on_gpu<float>(A, B);
  float *d_A = std::get<0>(dptrs);
  float *d_B = std::get<1>(dptrs);
  float *d_C = std::get<2>(dptrs);

  float milliseconds = 0.0;
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // loop over benchmark for this size
  for (auto _ : state) {
    hipEventRecord(start);
    fwgpu::cublas_sgemm(d_A, d_B, d_C, N, N, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);
    state.SetIterationTime(milliseconds / 1000);
  }

  double flops_per_itr = 2 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  // free device buffers
  fwgpu::internal::dealloc_gemm_mats_on_gpu<float>(dptrs);
}
BENCHMARK(BM_CublasSgemm)->RangeMultiplier(2)->Range(64, 4096)->UseManualTime();
