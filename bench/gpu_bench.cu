#include <tuple>

#include "benchmark/benchmark.h"
#include "fwgpu/Matrix.hpp"
#include "fwgpu/gpu_gemm.cuh"
#include "fwgpu/gpu_srgemm.cuh"
#include "fwgpu/gpu_srgemm.hpp"
#include "fwgpu/internal/utils.cuh"


static void BM_GpuGemmCutlass(benchmark::State &state) {
  const auto N = state.range(0);

  // init input matrices for this benchmark size N
  auto A = fwgpu::Matrix<float>(N, N, 1.5f);
  auto B = fwgpu::Matrix<float>(N, N, 1.5f);
  auto C = fwgpu::Matrix<float>(N, N, 0.0f);

  // allocate device buffers
  auto dptrs = fwgpu::internal::alloc_and_init_device_gemm_mats<float>(A, B, C);
  float *d_A = std::get<0>(dptrs);
  float *d_B = std::get<1>(dptrs);
  float *d_C = std::get<2>(dptrs);

  float milliseconds = 0.0;
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // loop over benchmark for this size
  for (auto _ : state) {
    hipEventRecord(start);
    fwgpu::cutlass_sgemm_nn(N, N, N, 1.0, d_A, N, d_B, N, 0.0, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);
    state.SetIterationTime(milliseconds / 1000);
  }

  double flops_per_itr = 2 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  // free device buffers
  fwgpu::internal::dealloc_device_gemm_mats<float>(dptrs);
}
BENCHMARK(BM_GpuGemmCutlass)->RangeMultiplier(2)->Range(64, 4096)->UseManualTime();

static void BM_GpuSrgemmCutlass(benchmark::State &state) {
  const auto N = state.range(0);

  // init input matrices for this benchmark size N
  auto A = fwgpu::Matrix<float>(N, N, 1.5f);
  auto B = fwgpu::Matrix<float>(N, N, 1.5f);
  auto C = fwgpu::Matrix<float>(N, N, 0.0f);

  // allocate device buffers
  auto dptrs = fwgpu::internal::alloc_and_init_device_gemm_mats<float>(A, B, C);
  float *d_A = std::get<0>(dptrs);
  float *d_B = std::get<1>(dptrs);
  float *d_C = std::get<2>(dptrs);

  float milliseconds = 0.0;
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // loop over benchmark for this size
  for (auto _ : state) {
    hipEventRecord(start);
    fwgpu::cutlass_srsgemm_nn(N, N, N, d_A, N, d_B, N, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);
    state.SetIterationTime(milliseconds / 1000);
  }

  double flops_per_itr = 2 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  // free device buffers
  fwgpu::internal::dealloc_device_gemm_mats<float>(dptrs);
}
BENCHMARK(BM_GpuSrgemmCutlass)->RangeMultiplier(2)->Range(64, 4096)->UseManualTime();

static void BM_GpuSrgemmCutlass_TS_Outer(benchmark::State &state) {
  const auto N = state.range(0);

  auto m = N;
  auto n = N;
  auto k = N / 8;

  // init input matrices for this benchmark size N
  auto A = fwgpu::Matrix<float>(m, k, 1.5f);
  auto B = fwgpu::Matrix<float>(k, n, 1.5f);
  auto C = fwgpu::Matrix<float>(m, n, 0.0f);

  // allocate device buffers
  auto dptrs = fwgpu::internal::alloc_and_init_device_gemm_mats<float>(A, B, C);
  float *d_A = std::get<0>(dptrs);
  float *d_B = std::get<1>(dptrs);
  float *d_C = std::get<2>(dptrs);

  float milliseconds = 0.0;
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // loop over benchmark for this size
  for (auto _ : state) {
    hipEventRecord(start);
    fwgpu::cutlass_srsgemm_nn(m, n, k, d_A, m, d_B, k, d_C, m, true);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);
    state.SetIterationTime(milliseconds / 1000);
  }

  double flops_per_itr = 2 * N * N * (N / 4);
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  // free device buffers
  fwgpu::internal::dealloc_device_gemm_mats<float>(dptrs);
}
BENCHMARK(BM_GpuSrgemmCutlass_TS_Outer)
    ->RangeMultiplier(2)
    ->Range(64, 4096)
    ->UseManualTime();

static void BM_GpuSrgemmCutlass_TS_Inner(benchmark::State &state) {
  const auto N = state.range(0);

  auto m = N / 8;
  auto n = N / 8;
  auto k = N;

  // init input matrices for this benchmark size N
  auto A = fwgpu::Matrix<float>(m, k, 1.5f);
  auto B = fwgpu::Matrix<float>(k, n, 1.5f);
  auto C = fwgpu::Matrix<float>(m, n, 0.0f);

  // allocate device buffers
  auto dptrs = fwgpu::internal::alloc_and_init_device_gemm_mats<float>(A, B, C);
  float *d_A = std::get<0>(dptrs);
  float *d_B = std::get<1>(dptrs);
  float *d_C = std::get<2>(dptrs);

  float milliseconds = 0.0;
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // loop over benchmark for this size
  for (auto _ : state) {
    hipEventRecord(start);
    fwgpu::cutlass_srsgemm_nn(m, n, k, d_A, m, d_B, k, d_C, m, true);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);
    state.SetIterationTime(milliseconds / 1000);
  }

  double flops_per_itr = 2 * (N / 4) * (N / 4) * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  // free device buffers
  fwgpu::internal::dealloc_device_gemm_mats<float>(dptrs);
}
BENCHMARK(BM_GpuSrgemmCutlass_TS_Inner)
    ->RangeMultiplier(2)
    ->Range(64, 4096)
    ->UseManualTime();

static void BM_GpuSrgemmNaive(benchmark::State &state) {
  const auto N = state.range(0);

  // init input matrices for this benchmark size N
  auto A = fwgpu::Matrix<float>(N, N, 1.5f);
  auto B = fwgpu::Matrix<float>(N, N, 1.5f);
  auto C = fwgpu::Matrix<float>(N, N, 0.0f);

  // allocate device buffers
  auto dptrs = fwgpu::internal::alloc_and_init_device_gemm_mats<float>(A, B, C);
  float *d_A = std::get<0>(dptrs);
  float *d_B = std::get<1>(dptrs);
  float *d_C = std::get<2>(dptrs);

  float milliseconds = 0.0;
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // loop over benchmark for this size
  dim3 threads(16, 16);
  dim3 blocks((N - 1) / 16 + 1, (N - 1) / 16 + 1);
  for (auto _ : state) {
    hipEventRecord(start);
    fwgpu::gpu_srgemm_naive<float><<<blocks, threads>>>(N, N, N, d_A, N, d_B, N, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);
    state.SetIterationTime(milliseconds / 1000);
  }

  double flops_per_itr = 2 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  // free device buffers
  fwgpu::internal::dealloc_device_gemm_mats<float>(dptrs);
}
BENCHMARK(BM_GpuSrgemmNaive)->RangeMultiplier(2)->Range(64, 4096)->UseManualTime();

static void BM_GpuGemmNaive(benchmark::State &state) {
  const auto N = state.range(0);

  // init input matrices for this benchmark size N
  auto A = fwgpu::Matrix<float>(N, N, 1.5f);
  auto B = fwgpu::Matrix<float>(N, N, 1.5f);
  auto C = fwgpu::Matrix<float>(N, N, 0.0f);

  // allocate device buffers
  auto dptrs = fwgpu::internal::alloc_and_init_device_gemm_mats<float>(A, B, C);
  float *d_A = std::get<0>(dptrs);
  float *d_B = std::get<1>(dptrs);
  float *d_C = std::get<2>(dptrs);

  float milliseconds = 0.0;
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // loop over benchmark for this size
  dim3 threads(16, 16);
  dim3 blocks((N - 1) / 16 + 1, (N - 1) / 16 + 1);
  for (auto _ : state) {
    hipEventRecord(start);
    fwgpu::gpu_gemm_naive<float><<<blocks, threads>>>(N, N, N, d_A, d_B, d_C);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);
    state.SetIterationTime(milliseconds / 1000);
  }

  double flops_per_itr = 2 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  // free device buffers
  fwgpu::internal::dealloc_device_gemm_mats<float>(dptrs);
}
BENCHMARK(BM_GpuGemmNaive)->RangeMultiplier(2)->Range(64, 4096)->UseManualTime();

static void BM_CublasSgemm(benchmark::State &state) {
  const auto N = state.range(0);

  // init input matrices for this benchmark size N
  auto A = fwgpu::Matrix<float>(N, N, 1.5f);
  auto B = fwgpu::Matrix<float>(N, N, 1.5f);
  auto C = fwgpu::Matrix<float>(N, N, 0.0f);

  // allocate device buffers
  auto dptrs = fwgpu::internal::alloc_and_init_device_gemm_mats<float>(A, B, C);
  float *d_A = std::get<0>(dptrs);
  float *d_B = std::get<1>(dptrs);
  float *d_C = std::get<2>(dptrs);

  float milliseconds = 0.0;
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // loop over benchmark for this size
  for (auto _ : state) {
    hipEventRecord(start);
    fwgpu::cublas_sgemm(d_A, d_B, d_C, N, N, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);
    state.SetIterationTime(milliseconds / 1000);
  }

  double flops_per_itr = 2 * N * N * N;
  state.counters["Flop/s"]
      = benchmark::Counter(flops_per_itr, benchmark::Counter::kIsIterationInvariantRate);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  // free device buffers
  fwgpu::internal::dealloc_device_gemm_mats<float>(dptrs);
}
BENCHMARK(BM_CublasSgemm)->RangeMultiplier(2)->Range(64, 4096)->UseManualTime();
