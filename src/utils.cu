#include "fwgpu/utils.hpp"

namespace fwgpu {

auto malloc_device(void **dptr, size_t size) -> int {
  return static_cast<int>(hipMalloc(dptr, size));
}

auto free_device(void *dbuf) -> int { return static_cast<int>(hipFree(dbuf)); }

auto memcpy_d2h(void *dest, const void *src, size_t size) -> int {
  return static_cast<int>(hipMemcpy(dest, src, size, hipMemcpyDeviceToHost));
}

auto memcpy_h2d(void *dest, const void *src, size_t size) -> int {
  return static_cast<int>(hipMemcpy(dest, src, size, hipMemcpyHostToDevice));
}

auto memcpy_2d_h2d(
    void *dest,
    size_t dpitch,
    const void *src,
    size_t spitch,
    size_t width,
    size_t height) -> int {
  return static_cast<int>(
      hipMemcpy2D(dest, dpitch, src, spitch, width, height, hipMemcpyHostToDevice));

}

auto memcpy_2d_d2h(
    void *dest,
    size_t dpitch,
    const void *src,
    size_t spitch,
    size_t width,
    size_t height) -> int {
  return static_cast<int>(
      hipMemcpy2D(dest, dpitch, src, spitch, width, height, hipMemcpyDeviceToHost));
}

} // namespace fwgpu
