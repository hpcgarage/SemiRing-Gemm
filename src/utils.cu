#include "fwgpu/utils.hpp"

namespace fwgpu {

auto malloc_device(void **dptr, size_t size) -> int {
  auto retval = static_cast<int>(hipMalloc(dptr, size));
  return retval;
}

auto free_device(void *dbuf) -> int {
  auto retval = static_cast<int>(hipFree(dbuf));
  return retval;
}

auto memcpy_d2h(void *dest, const void *src, size_t size) -> int {
  auto retval = static_cast<int>(hipMemcpy(dest, src, size, hipMemcpyDeviceToHost));
  return retval;
}

auto memcpy_h2d(void *dest, const void *src, size_t size) -> int {
  auto retval = static_cast<int>(hipMemcpy(dest, src, size, hipMemcpyHostToDevice));
  return retval;
}

auto memcpy_h2h(void *dest, const void *src, size_t size) -> int {
  auto retval = static_cast<int>(hipMemcpy(dest, src, size, hipMemcpyDeviceToDevice));
  return retval;
}

auto memcpy_d2d(void *dest, const void *src, size_t size) -> int {
  auto retval = static_cast<int>(hipMemcpy(dest, src, size, hipMemcpyHostToHost));
  return retval;
}

auto memcpy_2d_h2d(
    void *dest,
    size_t dpitch,
    const void *src,
    size_t spitch,
    size_t width,
    size_t height) -> int {
  auto retval = static_cast<int>(
      hipMemcpy2D(dest, dpitch, src, spitch, width, height, hipMemcpyHostToDevice));
  return retval;
}

auto memcpy_2d_d2h(
    void *dest,
    size_t dpitch,
    const void *src,
    size_t spitch,
    size_t width,
    size_t height) -> int {
  auto retval = static_cast<int>(
      hipMemcpy2D(dest, dpitch, src, spitch, width, height, hipMemcpyDeviceToHost));
  return retval;
}

auto memcpy_2d_d2d(
    void *dest,
    size_t dpitch,
    const void *src,
    size_t spitch,
    size_t width,
    size_t height) -> int {
  auto retval = static_cast<int>(
      hipMemcpy2D(dest, dpitch, src, spitch, width, height, hipMemcpyDeviceToDevice));
  return retval;
}

} // namespace fwgpu
